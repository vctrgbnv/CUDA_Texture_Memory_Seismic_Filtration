#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(err) do { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Sequential median filter
void median_filter_seq(const float* input, float* output, int M, int N, int W) {
    int radius = (W - 1) / 2;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            std::vector<float> window;
            for (int x = std::max(0, i - radius); x <= std::min(M - 1, i + radius); ++x) {
                for (int y = std::max(0, j - radius); y <= std::min(N - 1, j + radius); ++y) {
                    window.push_back(input[x * N + y]);
                }
            }
            std::sort(window.begin(), window.end());
            output[i * N + j] = window[window.size() / 2];
        }
    }
}

// CUDA kernel without Texture Memory
__global__ void median_filter_kernel_global(const float* d_input, float* d_output, int M, int N, int W) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M && j < N) {
        const int MAX_WINDOW_SIZE = 49; // Supports W up to 7
        float window[MAX_WINDOW_SIZE];
        int radius = (W - 1) / 2;
        int row_start = max(0, i - radius);
        int row_end = min(M - 1, i + radius);
        int col_start = max(0, j - radius);
        int col_end = min(N - 1, j + radius);
        int count = 0;
        for (int x = row_start; x <= row_end; ++x) {
            for (int y = col_start; y <= col_end; ++y) {
                window[count++] = d_input[x * N + y];
            }
        }
        // Bubble sort
        for (int k = 0; k < count - 1; ++k) {
            for (int l = 0; l < count - 1 - k; ++l) {
                if (window[l] > window[l + 1]) {
                    float temp = window[l];
                    window[l] = window[l + 1];
                    window[l + 1] = temp;
                }
            }
        }
        d_output[i * N + j] = window[count / 2];
    }
}

// CUDA kernel with Texture Memory
__global__ void median_filter_kernel_texture(hipTextureObject_t texObj, float* d_output, int M, int N, int W) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M && j < N) {
        const int MAX_WINDOW_SIZE = 49; // Supports W up to 7
        float window[MAX_WINDOW_SIZE];
        int radius = (W - 1) / 2;
        int row_start = max(0, i - radius);
        int row_end = min(M - 1, i + radius);
        int col_start = max(0, j - radius);
        int col_end = min(N - 1, j + radius);
        int count = 0;
        for (int x = row_start; x <= row_end; ++x) {
            for (int y = col_start; y <= col_end; ++y) {
                window[count++] = tex2D<float>(texObj, y, x);
            }
        }
        // Bubble sort
        for (int k = 0; k < count - 1; ++k) {
            for (int l = 0; l < count - 1 - k; ++l) {
                if (window[l] > window[l + 1]) {
                    float temp = window[l];
                    window[l] = window[l + 1];
                    window[l + 1] = temp;
                }
            }
        }
        d_output[i * N + j] = window[count / 2];
    }
}

// Function to generate synthetic seismic data
void generate_synthetic_data(float* data, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            // Add horizontal events every 100 rows, else random noise
            data[i * N + j] = (i % 100 == 0) ? 1.0f : (rand() % 1000) / 1000.0f - 0.5f;
        }
    }
}

// Function to save data to binary file
void save_data(const float* data, int M, int N, const std::string& filename) {
    std::ofstream file(filename, std::ios::binary);
    file.write(reinterpret_cast<const char*>(&M), sizeof(int));
    file.write(reinterpret_cast<const char*>(&N), sizeof(int));
    file.write(reinterpret_cast<const char*>(data), M * N * sizeof(float));
    file.close();
}

int main() {
    // Определяем размеры данных
    const int N = 4301;
    const int M = 600;
    const size_t num_elements = static_cast<size_t>(M) * N;

    // Создаем вектор для хранения данных
    std::vector<float> h_input(num_elements);

    // Открываем бинарный файл
    std::ifstream file("one_SP.bin", std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Ошибка: не удалось открыть файл one_SP.bin" << std::endl;
        return 1;
    }

    // Проверяем размер файла
    file.seekg(0, std::ios::end);
    size_t file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    size_t expected_size = num_elements * sizeof(double);
    if (file_size != expected_size) {
        std::cerr << "Ошибка: размер файла " << file_size
            << " не соответствует ожидаемому " << expected_size << std::endl;
        file.close();
        return 1;
    }

    // Создаем буфер для чтения данных типа double
    std::vector<double> buffer(num_elements);

    // Читаем данные из файла в буфер
    file.read(reinterpret_cast<char*>(buffer.data()), expected_size);
    if (!file) {
        std::cerr << "Ошибка: не удалось прочитать данные из файла" << std::endl;
        file.close();
        return 1;
    }

    // Закрываем файл
    file.close();

    // Преобразуем данные из double в float
    std::transform(buffer.begin(), buffer.end(), h_input.begin(),
        [](double d) { return static_cast<float>(d); });
    
//    const int M = 512; // Rows (timestep)
//    const int N = 512; // Columns (offset)
    //const int window_sizes[] = { 3, 5, 7};
    const int window_sizes[] = { 7 };
    const int block_sizes[] = { 16 };
    const int num_windows = 1;
    const int num_blocks = 1;

    // Allocate host memory
   // std::vector<float> h_input(M * N);
    std::vector<float> h_output_seq(M * N);
    std::vector<float> h_output_gpu(M * N);
    std::vector<float> h_output_tex(M * N);


    save_data(h_input.data(), M, N, "input_seismic.dat");

    // Task 1: Sequential Implementation
    std::cout << "Sequential Execution Times (seconds):\n";
    for (int w = 0; w < num_windows; ++w) {
        int W = window_sizes[w];
        auto start = std::chrono::high_resolution_clock::now();
        median_filter_seq(h_input.data(), h_output_seq.data(), M, N, W);
        auto end = std::chrono::high_resolution_clock::now();
        double time = std::chrono::duration<double>(end - start).count();
        std::cout << "W=" << W << ": " << time << "\n";
        if (W == 3) save_data(h_output_seq.data(), M, N, "output_seq_w3.dat");
    }
    std::cout << "\n";

    // Allocate device memory for GPU versions
    float* d_input, * d_output;
    CHECK_CUDA_ERROR(hipMalloc(&d_input, M * N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, M * N * sizeof(float)));

    // Task 2: GPU Implementation without Texture Memory
    std::cout << "GPU Execution Times without Texture Memory (seconds):\n";
    for (int w = 0; w < num_windows; ++w) {
        int W = window_sizes[w];
        std::cout << "W=" << W << ":\n";
        for (int b = 0; b < num_blocks; ++b) {
            int block_size = block_sizes[b];
            dim3 blockDim(block_size, block_size);
            dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

            hipEvent_t start, stop;
            CHECK_CUDA_ERROR(hipEventCreate(&start));
            CHECK_CUDA_ERROR(hipEventCreate(&stop));
            CHECK_CUDA_ERROR(hipEventRecord(start));

            CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
            median_filter_kernel_global << <gridDim, blockDim >> > (d_input, d_output, M, N, W);
            CHECK_CUDA_ERROR(hipGetLastError());
            CHECK_CUDA_ERROR(hipMemcpy(h_output_gpu.data(), d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));

            CHECK_CUDA_ERROR(hipEventRecord(stop));
            CHECK_CUDA_ERROR(hipEventSynchronize(stop));
            float time_ms;
            CHECK_CUDA_ERROR(hipEventElapsedTime(&time_ms, start, stop));
            std::cout << "  Block=" << block_size << "x" << block_size << ": " << time_ms / 1000.0 << "\n";

            CHECK_CUDA_ERROR(hipEventDestroy(start));
            CHECK_CUDA_ERROR(hipEventDestroy(stop));

            if (W == 7 && block_size == 16) save_data(h_output_gpu.data(), M, N, "output_gpu_w7_b16.dat");
        }
    }
    std::cout << "\n";

    // Task 3: GPU Implementation with Texture Memory
    hipArray* cuArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    CHECK_CUDA_ERROR(hipMallocArray(&cuArray, &channelDesc, N, M));
    CHECK_CUDA_ERROR(hipMemcpy2DToArray(cuArray, 0, 0, h_input.data(), N * sizeof(float), N * sizeof(float), M, hipMemcpyHostToDevice));

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    hipTextureObject_t texObj;
    CHECK_CUDA_ERROR(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

    std::cout << "GPU Execution Times with Texture Memory (seconds):\n";
    for (int w = 0; w < num_windows; ++w) {
        int W = window_sizes[w];
        std::cout << "W=" << W << ":\n";
        for (int b = 0; b < num_blocks; ++b) {
            int block_size = block_sizes[b];
            dim3 blockDim(block_size, block_size);
            dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

            hipEvent_t start, stop;
            CHECK_CUDA_ERROR(hipEventCreate(&start));
            CHECK_CUDA_ERROR(hipEventCreate(&stop));
            CHECK_CUDA_ERROR(hipEventRecord(start));

            median_filter_kernel_texture << <gridDim, blockDim >> > (texObj, d_output, M, N, W);
            CHECK_CUDA_ERROR(hipGetLastError());
            CHECK_CUDA_ERROR(hipMemcpy(h_output_tex.data(), d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));

            CHECK_CUDA_ERROR(hipEventRecord(stop));
            CHECK_CUDA_ERROR(hipEventSynchronize(stop));
            float time_ms;
            CHECK_CUDA_ERROR(hipEventElapsedTime(&time_ms, start, stop));
            std::cout << "  Block=" << block_size << "x" << block_size << ": " << time_ms / 1000.0 << "\n";

            CHECK_CUDA_ERROR(hipEventDestroy(start));
            CHECK_CUDA_ERROR(hipEventDestroy(stop));

            if (W == 3 && block_size == 16) save_data(h_output_tex.data(), M, N, "output_tex_w3_b16.dat");
        }
    }
    std::cout << "\n";

    // Task 4: Performance Analysis for W=5, Block=16
    int W = 5;
    int block_size = 16;
    dim3 blockDim(block_size, block_size);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord(start));

    median_filter_kernel_texture << <gridDim, blockDim >> > (texObj, d_output, M, N, W);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipMemcpy(h_output_tex.data(), d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float time_ms;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time_ms, start, stop));
    double time_s = time_ms / 1000.0;

    // Effective Memory Bandwidth
    size_t reads = M * N * W * W * sizeof(float); // Each thread reads W*W floats
    size_t writes = M * N * sizeof(float);        // Each thread writes 1 float
    size_t total_bytes = reads + writes;
    double bandwidth = total_bytes / time_s / 1e9; // GB/s
    std::cout << "Effective Memory Bandwidth (W=5, Block=16): " << bandwidth << " GB/s\n";
    // Note: Compare with GPU's theoretical bandwidth (e.g., for NVIDIA GTX 1080, ~320 GB/s)

    // Effective Computational Throughput (approximation based on bubble sort operations)
    size_t comparisons = M * N * (W * W) * (W * W - 1) / 2; // Approx. bubble sort comparisons
    double throughput = comparisons / time_s / 1e9; // GFLOPS
    std::cout << "Effective Computational Throughput (W=5, Block=16): " << throughput << " GFLOPS\n";
    // Note: Compare with GPU's theoretical FLOPS (e.g., for GTX 1080, ~9 TFLOPS)

    // Cleanup
    CHECK_CUDA_ERROR(hipDestroyTextureObject(texObj));
    CHECK_CUDA_ERROR(hipFreeArray(cuArray));
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_output));

    //// для проверки можно вывести первые несколько элементов (опционально)
    //for (int i = 0; i < 5 && i < num_elements; ++i) {
    //    std::cout << "h_input[" << i << "] = " << h_input[i] << std::endl;
    //}
    //// для проверки можно вывести первые несколько элементов (опционально)
    //for (int i = 0; i < 5 && i < num_elements; ++i) {
    //    std::cout << "h_output_seq[" << i << "] = " << h_output_seq[i] << std::endl;
    //}
    //// для проверки можно вывести первые несколько элементов (опционально)
    //for (int i = 0; i < 5 && i < num_elements; ++i) {
    //    std::cout << "h_output_gpu[" << i << "] = " << h_output_gpu[i] << std::endl;
    //}
    //// для проверки можно вывести первые несколько элементов (опционально)
    //for (int i = 0; i < 5 && i < num_elements; ++i) {
    //    std::cout << "h_output_tex[" << i << "] = " << h_output_tex[i] << std::endl;
    //}
    return 0;
}